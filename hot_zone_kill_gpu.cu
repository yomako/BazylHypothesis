#include "hip/hip_runtime.h"
#include "book.h"
#include "hot_zone_kill_gpu.h"
#define REPEATS 5000
#define N 10


struct BallsSet
{
	Ball balls[REPEATS][N];
};

__constant__ Params params;

__global__ void kernel(BallsSet *balls_from_host, int *results){
	__shared__ Ball balls[N];
	__shared__ bool running;
	
	int ball_id = threadIdx.x;
	enum State state = GAME_ON;
	double tick = 0.0;
	running = true;

	balls[ball_id].x = balls_from_host->balls[blockIdx.x][ball_id].x;
	balls[ball_id].y = balls_from_host->balls[blockIdx.x][ball_id].y;
	balls[ball_id].v_x = balls_from_host->balls[blockIdx.x][ball_id].v_x;
	balls[ball_id].v_y = balls_from_host->balls[blockIdx.x][ball_id].v_y;
	balls[ball_id].tick_base = balls_from_host->balls[blockIdx.x][ball_id].tick_base;
	__syncthreads();

	while (running) {
		__shared__ double lowests[N];
		__shared__ double lowest;
		__shared__ bool pocket_is_lowest;
		__shared__ bool pockets[N];
		__shared__ double min_putting_time;
		__shared__ double putting_out_id;
		shortcut_step_part1(balls, &params, ball_id, tick, &state, lowests, pockets);
		if(ball_id == 0){
			lowest = nan("1");
			state = GAME_ON;
			pocket_is_lowest = false;
			int lowest_id = 0;
			for(int i = 0; i < N; i++){
				if(is_lower(lowests[i], lowest)){
					lowest = lowests[i];
					lowest_id = i;
				}
			}
			pocket_is_lowest = pockets[lowest_id];
		}
		__syncthreads();
		if(params.motion_mode == UNIFORMLY_DECELERATED){
			__shared__ double min_putting_times[N];
			shortcut_step_part1d5(balls, &params, ball_id, lowest, tick, min_putting_times);
			if(ball_id == 0){
				min_putting_time = nan("1");
				putting_out_id = -1;
				for(int i = 0; i < N; i++){
					if(is_lower(min_putting_times[i], min_putting_time)){
						min_putting_time = min_putting_times[i];
						putting_out_id = i;
					}
				}
			}
		}
		__syncthreads();
		if(params.motion_mode == UNIFORMLY_DECELERATED && putting_out_id != -1){
			tick += ud_putting_out(balls, &params, putting_out_id, tick);
			continue;
		}
		tick += shortcut_step_part2(balls, &params, ball_id, lowest, pocket_is_lowest, tick, &state) * params.delta_t;
		if(ball_id == 0){
			if(isnan(tick)) state = check_table(balls, &params, tick);
			if(state == BALL_BEYOND_TABLE && running){
				running = false;
				(*results)++;
			}
			if(state == LACK_OF_ENERGY && running){
				running = false;
			}
		}
		__syncthreads();
		if(running){
			mechanics_step(balls, &params, ball_id, tick);
			tick += params.delta_t;
		}
		__syncthreads();
	}
	printf("%d\n", *results);
}

int main( void ) {
	hipEvent_t start, stop;
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventRecord( start, 0 ) );
	srand(2137); 
	bool running = true;
	Params pParams;
	pParams.width = 2*640;
	pParams.height = 2*480;
	pParams.a = 2*600;
	pParams.b = 2*400;
	pParams.R = 8.0;
	pParams.top_border = (pParams.height - pParams.b) / 2;
	pParams.bottom_border = (pParams.height + pParams.b) / 2;
	pParams.right_border = (pParams.width + pParams.a) / 2;
	pParams.left_border = (pParams.width - pParams.a) / 2;
	pParams.mu = 0.0002;
	pParams.l = 60.0;
	pParams.k = 1;
	pParams.delta_t = 0.001;
	pParams.v_max = 0.4*sqrt(2);
	pParams.n = N;
	pParams.motion_mode = PROPORTIONAL_TO_VELOCITY;
	//pParams.motion_mode = UNIFORMLY_DECELERATED;

	Ball balls[REPEATS][N];
	BallsSet pBallsset;
	BallsSet *ballsset;
	int pResults = 0;
	int *results;

	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(params), &pParams, sizeof(Params)));

	for(int i=0; i < REPEATS; i++){
		balls_init(balls[i], &pParams);
		
	}
	for(int i=0; i < REPEATS; i++){
		memcpy(pBallsset.balls[i], balls[i], N * sizeof (Ball));
	}
	HANDLE_ERROR( hipMalloc( (void**)&ballsset, sizeof(BallsSet) ) );
	HANDLE_ERROR( hipMalloc( (void**)&results, sizeof(int) ) );
	HANDLE_ERROR( hipMemcpy( ballsset, &pBallsset, sizeof(BallsSet), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( results, &pResults, sizeof(int), hipMemcpyHostToDevice ) );
	
	kernel<<<REPEATS,N>>>(ballsset, results);

	HANDLE_ERROR( hipMemcpy( &pResults, results, sizeof(int), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	float   elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	printf("p(c)=%lf\n", (double) pResults / REPEATS);
	printf( "Time to generate:  %f s\n", elapsedTime / 1000);
	
	HANDLE_ERROR( hipFree( ballsset ) );
	HANDLE_ERROR( hipFree( results ) );
	return 0;
}